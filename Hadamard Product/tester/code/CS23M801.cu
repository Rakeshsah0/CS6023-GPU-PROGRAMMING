#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;


__global__
void CalculateHadamardProduct(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int row=i/N;
    int col=i%N;
    if (i < N * N) {
        A[row*N+col] *= B[col*N+row];
    }
}

__global__
void FindWeightMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    int i=blockIdx.x*blockDim.x*blockDim.y + threadIdx.x*blockDim.y+threadIdx.y;
    if (i < N * N) {
        A[i] = (A[i] > B[i]) ? A[i] : B[i];
    }
}

__global__
void CalculateFinalMatrix(long int* A, long int* B, int N) {

    // TODO: Write your kernel here
    int i=blockIdx.x*gridDim.y*blockDim.x*blockDim.y + blockIdx.y*blockDim.x*blockDim.y+threadIdx.x*blockDim.y+threadIdx.y;
    int row = (i/(2*N))%N;
    int col = i%(N);
    if(i<4*N*N){
        B[i]*=A[(row*N+col)];
    }
}


int main(int argc, char** argv) {


    int N;
    cin >> N;
    long int* A = new long int[N * N];
    long int* B = new long int[N * N];
    long int* C = new long int[N * N];
    long int* D = new long int[2 * N * 2 * N];


    for (long int i = 0; i < N * N; i++) {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++) {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++) {
        cin >> D[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
    */

    long int* d_A;
    long int* d_B;
    long int* d_C;
    long int* d_D;

    hipMalloc((void**)&d_A, N * N * sizeof(long int));
    hipMalloc((void**)&d_B, N * N * sizeof(long int));
    hipMalloc((void**)&d_C, N * N * sizeof(long int));
    hipMalloc((void**)&d_D, 2 * N * 2 * N * sizeof(long int));

    hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, 4*N * N * sizeof(long int), hipMemcpyHostToDevice);


    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);


    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;


    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);


    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;


    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);


    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    // Make sure your final output from the device is stored in d_D.

    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
    */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < 2 * N; i++) {
            for (long int j = 0; j < 2 * N; j++) {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}